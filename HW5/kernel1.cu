#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16

__global__ void mandelKernel(int resX, int resY, float stepX, float stepY, int *GPUresult, float lowerX, float lowerY, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    if(thisX < resX && thisY < resY){
        float x = lowerX + thisX * stepX;
        float y = lowerY + thisY * stepY;
        float z_re = x, z_im = y;
        int i;
        for (i = 0; i < maxIterations; ++i){
            if (z_re * z_re + z_im * z_im > 4.f)
                break;
            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = x + new_re;
            z_im = y + new_im;
        }
        GPUresult[thisY * resX + thisX] = i;
    }
    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY;
    int *result_h = (int *)malloc(size * sizeof(int));
    int *result_d;
    hipMalloc((void **)&result_d, size * sizeof(int));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock((resX + BLOCK_SIZE - 1) / BLOCK_SIZE, (resY + BLOCK_SIZE - 1) / BLOCK_SIZE);

    mandelKernel<<<numBlock, blockSize>>>(resX, resY, stepX, stepY, result_d, lowerX, lowerY, maxIterations);

    hipDeviceSynchronize();
    hipMemcpy(result_h, result_d, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(img, result_h, size * sizeof(int), hipMemcpyHostToHost);

    hipFree(result_d);
    free(result_h);
    
}
