#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 8
#define GROUP_SIZE 2

__global__ void mandelKernel(int resX, int resY, float stepX, float stepY, int *GPUresult, float lowerX, float lowerY, int maxIterations, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y);
    float x;
    float y;
    float z_re, z_im;

    for(int i = 0; i < GROUP_SIZE; i++){
        if(thisX + i >= resX || thisY >= resY)
            continue;
        x = lowerX + (thisX+i) * stepX;
        y = lowerY + thisY * stepY;
        z_re = x, z_im = y;
        int t = 0;
        for (t = 0; t < maxIterations; ++t){
            if (z_re * z_re + z_im * z_im > 4.f)
                break;
            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = x + new_re;
            z_im = y + new_im;
        }
        *((int*)((char*)GPUresult + thisY * pitch) + thisX + i) = t;
    }    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    size_t pitch = 0;

    int size = resX * resY;
    int *result_h;
    int *result_d;
    hipMallocPitch((void **)&result_d, &pitch, resX * sizeof(int), resY);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock((resX + GROUP_SIZE * BLOCK_SIZE - 1) / BLOCK_SIZE / GROUP_SIZE, (resY +  GROUP_SIZE * BLOCK_SIZE - 1) / BLOCK_SIZE);

    mandelKernel<<<numBlock, blockSize>>>(resX, resY, stepX, stepY, result_d, lowerX, lowerY, maxIterations, pitch);

    hipDeviceSynchronize();
    hipMemcpy2D(img, resX * sizeof(int), result_d, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

    hipFree(result_d);
}
